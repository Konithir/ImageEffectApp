#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

using namespace std;

extern "C" { 
	__declspec(dllexport) void Detect() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	printf("Number of CUDA Devices: %d\n", nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		printf("Number of multiprocessors:     %d\n", prop.multiProcessorCount);
	}
}
}

struct Rgb {
	unsigned char r, g, b;
};

__global__ void ProcessPixel_Negatyw(unsigned char* d_bitmapaWe, unsigned char* d_bitmapaWy, unsigned long int* d_size)
{
	unsigned long int id = threadIdx.x + blockIdx.x * 256;

	if (id > *d_size)
		return;

		d_bitmapaWy[id] = 255 - d_bitmapaWe[id];
}
__global__ void ProcessPixel_Jasnosc(unsigned char* d_bitmapaWe, unsigned char* d_bitmapaWy, unsigned long int* d_size)
{
	unsigned long int id = threadIdx.x + blockIdx.x * 256;

	if (id > *d_size)
		return;
	if (id % 3 != 1)
		return;
	unsigned char suma = (0.299*d_bitmapaWe[id-1] + 0.587*d_bitmapaWe[id] + 0.114*d_bitmapaWe[id+1]);
	d_bitmapaWy[id] = suma;
	d_bitmapaWy[id+1] = suma;
	d_bitmapaWy[id-1] = suma;
}
__global__ void ProcessPixel_Rozjasnianie(unsigned char* d_bitmapaWe, unsigned char* d_bitmapaWy, unsigned long int* d_size,int* d_wartosc)
{
	unsigned long int id = threadIdx.x + blockIdx.x * 256;

	if (id > *d_size)
		return;
	int suma = ((int)d_bitmapaWe[id] + *d_wartosc);
	if (suma > 255)
		suma = 255;
	if (suma < 0)
		suma = 0;
	d_bitmapaWy[id] = suma;
}

extern "C" {
	__declspec(dllexport) unsigned char* C_Negatyw(unsigned char* h_bitmapa, unsigned long int h_size)
	{
		
			unsigned char *d_bitmapaWe;
			unsigned char *d_bitmapaWy;

			unsigned long int *d_size = NULL;
			hipMalloc((void**)&d_size, sizeof(unsigned long int));
			hipMalloc((void**)&d_bitmapaWe, sizeof(unsigned char)* h_size);
			hipMalloc((void**)&d_bitmapaWy, sizeof(unsigned char)* h_size);
			hipMemcpy(d_bitmapaWe, h_bitmapa, h_size * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpy(d_size, &h_size, sizeof(unsigned long int), hipMemcpyHostToDevice);
			int blocks = (h_size / 256)+1;
			ProcessPixel_Negatyw << <blocks, 256 >> > (d_bitmapaWe, d_bitmapaWy, d_size);

			hipDeviceSynchronize();
			hipMemcpy(h_bitmapa, d_bitmapaWy, h_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

			hipFree(d_bitmapaWe);
			hipFree(d_bitmapaWy);
			hipFree(d_size);
			return h_bitmapa;
			
	}
}
extern "C" {
	__declspec(dllexport) unsigned char* C_Jasnosc(unsigned char* h_bitmapa, unsigned long int h_size)
	{

		unsigned char *d_bitmapaWe;
		unsigned char *d_bitmapaWy;

		unsigned long int *d_size = NULL;
		hipMalloc((void**)&d_size, sizeof(unsigned long int));
		hipMalloc((void**)&d_bitmapaWe, sizeof(unsigned char)* h_size);
		hipMalloc((void**)&d_bitmapaWy, sizeof(unsigned char)* h_size);
		hipMemcpy(d_bitmapaWe, h_bitmapa, h_size * sizeof(unsigned char), hipMemcpyHostToDevice);
		hipMemcpy(d_size, &h_size, sizeof(unsigned long int), hipMemcpyHostToDevice);
		int blocks = (h_size / 256) + 1;
		ProcessPixel_Jasnosc << <blocks, 256 >> > (d_bitmapaWe, d_bitmapaWy, d_size);

		hipDeviceSynchronize();
		hipMemcpy(h_bitmapa, d_bitmapaWy, h_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(d_bitmapaWe);
		hipFree(d_bitmapaWy);
		hipFree(d_size);
		return h_bitmapa;

	}
}
extern "C" {
	__declspec(dllexport) unsigned char* C_Rozjasnianie(unsigned char* h_bitmapa, unsigned long int h_size,int h_wartosc)
	{

		unsigned char *d_bitmapaWe;
		unsigned char *d_bitmapaWy;
		int *d_wartosc=NULL;
		unsigned long int *d_size = NULL;

		hipMalloc((void**)&d_size, sizeof(unsigned long int));
		hipMalloc((void**)&d_wartosc, sizeof(int));
		hipMalloc((void**)&d_bitmapaWe, sizeof(unsigned char)* h_size);
		hipMalloc((void**)&d_bitmapaWy, sizeof(unsigned char)* h_size);

		hipMemcpy(d_bitmapaWe, h_bitmapa, h_size * sizeof(unsigned char), hipMemcpyHostToDevice);
		hipMemcpy(d_size, &h_size, sizeof(unsigned long int), hipMemcpyHostToDevice);
		hipMemcpy(d_wartosc, &h_wartosc, sizeof(int), hipMemcpyHostToDevice);

		int blocks = (h_size / 256) + 1;
		ProcessPixel_Rozjasnianie << <blocks, 256 >> > (d_bitmapaWe, d_bitmapaWy, d_size,d_wartosc);

		hipDeviceSynchronize();
		hipMemcpy(h_bitmapa, d_bitmapaWy, h_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

		hipFree(d_bitmapaWe);
		hipFree(d_bitmapaWy);
		hipFree(d_size);
		return h_bitmapa;

	}
}

int main()
{	
	
	Detect();
	unsigned char Bitmapa[50] = {
		5,8,9,15,15,15,15,15,15,15, 
		15,15,15,15,15,15,15,15,15,15, 
		15,15,15,15,15,15,15,15,15,15, 
		15,15,15,15,15,15,15,15,15,15, 
		15,15,15,15,15,15,15,15,15,15, };

	for (int i = 0; i < 50; i++) {
		cout << Bitmapa[i];
	}

	C_Negatyw(Bitmapa, 50);
	for (int i = 0; i < 50; i++) {
		cout << Bitmapa[i];
	}
	
    return 0;
}